#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Fast R-CNN
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License [see fast-rcnn/LICENSE for details]
// Written by Ross Girshick
// Updated by Zheng Shou
// ------------------------------------------------------------------

#include <algorithm>
#include <cmath>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/io.hpp"

using std::max;

namespace caffe {

template <typename Dtype>
Dtype SmoothL1LossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {

    int count = bottom[0]->count();

  // Stable version of loss computation from input data
  const Dtype* input_data = bottom[0]->cpu_data();
  const Dtype* target = bottom[1]->cpu_data();
  caffe_sub(
      count,
      bottom[0]->cpu_data(),
      bottom[1]->cpu_data(),
      diff_.mutable_cpu_data());
  //LOG(INFO) << "diff_.mutable_cpu_data()[0]: "<<diff_.cpu_data()[0];
  //LOG(INFO) << "diff_.mutable_cpu_data()[1]: "<<diff_.cpu_data()[1];
  
  // has_weights_ is an option to be set in prototxt but we do not implement this
  // Alternatively, we set has_weights_ in smooth_L1_loss_layer.cpp to 1 if (bottom.size() == 3)
  // bottom[2]->cpu_data() are weights in hdf5 input for each reg class: # 2(N-1): 1 1 0 0 0 0 ... (N=4)
  // the reason to be "2"N-1 is there are 2 reg targets for each class. So this two should always be same as 1 or 0
  // We handle setting loss_weight_ and shuffle data when generating all segments/sliding windows
  // Here N doesn't include background class - class 0. so for binary cls, this is 0 0 (no reg) and 1 1 (do reg)
  if (has_weights_) {
    caffe_mul(
        count,
        bottom[2]->cpu_data(),
        diff_.cpu_data(),
        diff_.mutable_cpu_data());  // d := w * (b0 - b1)
  }	  
  
  // f(x) = 0.5 * x^2    if |x| < 1
  //        |x| - 0.5    otherwise
  Dtype loss = 0;
  for (int i = 0; i < count; ++i) {
    //LOG(INFO) << "input_data["<< i <<"]: "<< input_data[i];
    //LOG(INFO) << "target["<< i <<"]: "<< target[i];  
    if (abs(diff_.cpu_data()[i]) < 1) {
      loss += 0.5 * diff_.cpu_data()[i] * diff_.cpu_data()[i];
    } else {
      loss += abs(diff_.cpu_data()[i]) - 0.5;
    }
  }
  
  // loss_weight_ for smooth l1 loss layer
  loss = loss * loss_weight_;

  (*top)[0]->mutable_cpu_data()[0] = loss / bottom[0]->num();
  
  LOG(INFO) << "smooth loss: " << loss / bottom[0]->num();
  return loss / bottom[0]->num();
}

template <typename Dtype>
void SmoothL1LossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down, vector<Blob<Dtype>*>* bottom) {
	
  int count = diff_.count();
  
   for (int i = 0; i < count; ++i) {
  // f'(x) = x         if |x| < 1
  //       = sign(x)   otherwise
    if (abs(diff_.cpu_data()[i]) < 1) {
      diff_.mutable_cpu_data()[i] = diff_.cpu_data()[i];
    } else {
      diff_.mutable_cpu_data()[i] = (Dtype(0) < diff_.cpu_data()[i]) - (diff_.cpu_data()[i] < Dtype(0));
    }
  }
  
  for (int i = 0; i < 2; ++i) {
    if (i == 0) {
      const Dtype sign = (i == 0) ? 1 : -1;
	  // old version doesn't have "top[0]->cpu_diff()[0]"
      // loss_weight_ for smooth l1 loss layer
      const Dtype alpha = sign * loss_weight_ / (*bottom)[0]->num();
      caffe_cpu_axpby(
          (*bottom)[0]->count(),              // count
          alpha,                           // alpha
          diff_.cpu_data(),                // x
          Dtype(0),                        // beta
          (*bottom)[0]->mutable_cpu_diff());  // y
    }
  }
  
}

INSTANTIATE_CLASS(SmoothL1LossLayer);

}  // namespace caffe
